
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <vector>
#include <sstream>
#include <stdexcept>

inline void cuda_check_error(hipError_t error, const std::string filename, const std::size_t line, const std::string funcname, const std::string message = ""){
	if(error != hipSuccess){
		std::stringstream ss;
		ss << hipGetErrorString( error );
		if(message.length() != 0){
			ss << " : " << message;
		}
		ss << " [" << filename << ":" << line << " in " << funcname << "]";
		throw std::runtime_error(ss.str());
	}
}
#ifndef CUDA_CHECK_ERROR
#define CUDA_CHECK_ERROR(status) cuda_check_error(status, __FILE__, __LINE__, __func__)
#endif
#ifndef CUDA_CHECK_ERROR_M
#define CUDA_CHECK_ERROR_M(status, message) cuda_check_error(status, __FILE__, __LINE__, __func__, message)
#endif

template <class T>
__global__ void simple_copy_kernel(
    T* const dst_ptr,
    const T* const src_ptr,
    const std::size_t count
    ) {
  const auto tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= count) {
    return;
  }

  dst_ptr[tid] = src_ptr[tid];
}

void simple_copy(
    void* const dst_ptr,
    const void* const src_ptr,
    const std::size_t size,
    hipStream_t hip_stream = 0
    ) {
  const auto block_size = 1024;
  if (size % 16  == 0) {
    const auto count = size / 16;
    using data_t = ulong2;
    simple_copy_kernel<data_t><<<(count + block_size - 1) / block_size, block_size, 0, hip_stream>>>(reinterpret_cast<data_t*>(dst_ptr), reinterpret_cast<const data_t*>(src_ptr), count);
  } else if (size % 8 == 0) {
    const auto count = size / 8;
    using data_t = uint64_t;
    simple_copy_kernel<data_t><<<(count + block_size - 1) / block_size, block_size, 0, hip_stream>>>(reinterpret_cast<data_t*>(dst_ptr), reinterpret_cast<const data_t*>(src_ptr), count);
  } else if (size % 4 == 0) {
    const auto count = size / 4;
    using data_t = uint32_t;
    simple_copy_kernel<data_t><<<(count + block_size - 1) / block_size, block_size, 0, hip_stream>>>(reinterpret_cast<data_t*>(dst_ptr), reinterpret_cast<const data_t*>(src_ptr), count);
  } else if (size % 2 == 0) {
    const auto count = size / 2;
    using data_t = uint16_t;
    simple_copy_kernel<data_t><<<(count + block_size - 1) / block_size, block_size, 0, hip_stream>>>(reinterpret_cast<data_t*>(dst_ptr), reinterpret_cast<const data_t*>(src_ptr), count);
  } else {
    const auto count = size;
    using data_t = uint8_t;
    simple_copy_kernel<data_t><<<(count + block_size - 1) / block_size, block_size, 0, hip_stream>>>(reinterpret_cast<data_t*>(dst_ptr), reinterpret_cast<const data_t*>(src_ptr), count);
  }
}

template <class Func>
double measure_time(
    const std::size_t size,
    const Func func
    ) {
  void *dst_ptr, *src_ptr;
  hipMalloc(&dst_ptr, size);
  hipMalloc(&src_ptr, size);

  func(dst_ptr, src_ptr, size);

  hipDeviceSynchronize();
  const auto start_clock = std::chrono::system_clock::now();

  func(dst_ptr, src_ptr, size);

  hipDeviceSynchronize();
  const auto end_clock = std::chrono::system_clock::now();

  const auto elapsed_time = std::chrono::duration_cast<std::chrono::nanoseconds>(end_clock - start_clock).count() * 1e-9;

  hipFree(dst_ptr);
  hipFree(src_ptr);

  return elapsed_time;
}

int main() {
  std::printf("n,size_offset,size,cudaMemcpy_time,cudaMemcpy_bw,copy_kernel_time,copy_kernel_bw\n");

  for (const auto offset : std::vector<int>{0, -1, -2, 1, 2}) {
    for (std::uint32_t n = 0; n <= 32; n++) {
      const auto base_size = 1lu << n;
      if (offset < 0 && base_size <= -offset) continue;
      const auto size = base_size + offset;
      const auto cudaMemcpy_time  = measure_time(size + 1, [](void* const dst_ptr, const void* const src_ptr, const std::size_t size){hipMemcpy(dst_ptr, src_ptr, size, hipMemcpyDeviceToDevice);});
      const auto copy_kernel_time = measure_time(size + 1, [](void* const dst_ptr, const void* const src_ptr, const std::size_t size){simple_copy(dst_ptr, src_ptr, size);});
      std::printf(
          "%u,%d,%lu,%e,%e,%e,%e\n",
          n,
          offset,
          size,
          cudaMemcpy_time,
          size / cudaMemcpy_time,
          copy_kernel_time,
          size / copy_kernel_time
          );
      std::fflush(stdout);
    }
  }
}
